/*
============================================================================
Filename    : algorithm.c
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}


// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    
    double* d_input; double *d_output; //moi
    int length_2 = length*length;
	hipMalloc(&d_input, length_2 * sizeof(double)); //moi
	hipMalloc(&d_output, length_2 * sizeof(double)); //moi

    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
   
	hipMemcpy(d_input, input, length_2 * sizeof(double), hipMemcpyHostToDevice); //moi
	hipMemcpy(d_output, output, length_2 * sizeof(double), hipMemcpyHostToDevice); //moi
    
    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    //Copy array from host to device
    hipEventRecord(comp_start);
    /* GPU calculation goes here */
    
    int x_global = (blockIdx.x * blockDim.x) + threadIdx.x;//moi
    int y_global = (blockIdx.y * blockDim.y) + threadIdx.y;//moi
    
    for (int i = 0 ; i < iterations; i ++){
		
		if ((x_global == length/2 and y_global == length/2) or
			(x_global == length/2 - 1 and y_global == length/2) or
			(x_global == length/2 and y_global == length/2 - 1) or
			(x_global == length/2 - 1 and y_global == length/2 - 1)) {output[y_global * length + c_global] = 1000;} //moi
		
		if (x_global > 0 and x_global < length - 1 and y_global > 0 and y_global < length - 1) {
			
			output[(y_global)*(length)+(x_global)] = (input[(y_global-1)*(length)+(x_global-1)] +
												input[(y_global-1)*(length)+(x_global)]   +
												input[(y_global-1)*(length)+(x_global+1)] +
												input[(y_global)*(length)+(x_global-1)]   +
												input[(y_global)*(length)+(x_global)]     +
												input[(y_global)*(length)+(x_global+1)]   +
												input[(y_global+1)*(length)+(x_global-1)] +
												input[(y_global+1)*(length)+(x_global)]   +
												input[(y_global+1)*(length)+(x_global+1)] ) / 9;
		} //moi
	
		hipDeviceSynchronize();
	} //moi
		
    
    
    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
		
	hipMemcpy(d_input, input, length_2 * sizeof(double), hipMemcpyDeviceToHost); //moi
	hipMemcpy(d_output, output, length_2 * sizeof(double), hipMemcpyDeviceToHost); //moi
			
    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */
    
    hipFree(d_input); //moi
	hipFree(d_output); //moi
		

    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}
