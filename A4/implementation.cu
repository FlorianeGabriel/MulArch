#include "hip/hip_runtime.h"
/*
============================================================================
Filename    : algorithm.c
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void GPU_calculation(double*input, double* output, int lenght);

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}


// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    
    double* d_input; double *d_output; 
    int length_2 = length*length;
    
	if(hipMalloc(&d_input, length_2 * sizeof(double)) != hipSuccess){
		cout<<"error in hipMalloc"<<endl;
	} 
	if(hipMalloc(&d_output, length_2 * sizeof(double)) != hipSuccess){
		cout<<"error in hipMalloc"<<endl;
	} 
	
    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
    
	if(hipMemcpy(d_input, input, length_2 * sizeof(double), hipMemcpyHostToDevice) != hipSuccess){
		cout<<"error in hipMemcpy H2D"<<endl;
	}
	if(hipMemcpy(d_output, output, length_2 * sizeof(double), hipMemcpyHostToDevice) != hipSuccess){
		cout<<"error in hipMemcpy H2D"<<endl;
	} 
    
    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    //Copy array from host to device
    hipEventRecord(comp_start);
    /* GPU calculation goes here */
    
    for (int i = 0 ; i < iterations; i ++){
		
		GPU_calculation<<<length, length>>>(d_input, d_output, length);
	
		hipDeviceSynchronize();
		double* temp = d_input;
		d_input = d_output;
		d_output = temp; // Est ce que c'est nécessaire ??
			
	}
	
    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    	
	if(hipMemcpy(input, d_input, length_2 * sizeof(double), hipMemcpyDeviceToHost) != hipSuccess){
		cout<<"error in hipMemcpy D2H"<<endl;
	}
	if (hipMemcpy(output, d_output, length_2 * sizeof(double), hipMemcpyDeviceToHost) != hipSuccess){
		cout<<"error in hipMemcpy D2H"<<endl;
	}
		
    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */
    
    hipFree(d_input); 
	hipFree(d_output); 

    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}

__global__ void GPU_calculation(double* input, double* output, int length)
{
	int x_global = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y_global = (blockIdx.y * blockDim.y) + threadIdx.y;
		
	output[(y_global)*(length)+(x_global)] = (input[(y_global-1)*(length)+(x_global-1)] +
												input[(y_global-1)*(length)+(x_global)]   +
												input[(y_global-1)*(length)+(x_global+1)] +
												input[(y_global)*(length)+(x_global-1)]   +
												input[(y_global)*(length)+(x_global)]     +
												input[(y_global)*(length)+(x_global+1)]   +
												input[(y_global+1)*(length)+(x_global-1)] +
												input[(y_global+1)*(length)+(x_global)]   +
												input[(y_global+1)*(length)+(x_global+1)] ) / 9;
	
		
		/* center of the grid*/
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;
        
        /* border of the grid*/
        if(x_global == 0 or y_global == 0 or x_global == length - 1 or y_global == length - 1){
			output[(y_global)*(length)+(x_global)] = 0;
		}	
		
}
